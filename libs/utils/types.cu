/**
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 * Copyright (c) 2017, Daniel Thuerck, TU Darmstadt - GCC. All rights reserved.
 *
 * This software may be modified and distributed under the terms
 * of the BSD 3-clause license. See the LICENSE file for details.
 */

#include <libs/utils/types.cuh>
#include <libs/utils/types.impl.cuh>

NS_CULIP_BEGIN

/**
 * *****************************************************************************
 * ************************* TEMPLATE INSTANTIATIONS ***************************
 * *****************************************************************************
 */

template class dense_vector_t<char>;
template class dense_vector_t<mat_size_t>;

template class dense_vector_t<mat_int_t>;
template class col_major_matrix_t<mat_int_t>;
template class csr_matrix_t<mat_int_t>;
template class coo_matrix_t<mat_int_t>;

template class dense_vector_t<float>;
template class col_major_matrix_t<float>;
template class csr_matrix_t<float>;
template class coo_matrix_t<float>;

template class dense_vector_t<double>;
template class col_major_matrix_t<double>;
template class csr_matrix_t<double>;
template class coo_matrix_t<double>;

/* ************************************************************************** */

template
col_major_matrix_ptr<mat_int_t>
make_col_major_matrix_ptr(
    const mat_size_t _m,
    const mat_size_t _n,
    const bool _on_device);

template
col_major_matrix_ptr<float>
make_col_major_matrix_ptr(
    const mat_size_t _m,
    const mat_size_t _n,
    const bool _on_device);

template
col_major_matrix_ptr<double>
make_col_major_matrix_ptr(
    const mat_size_t _m,
    const mat_size_t _n,
    const bool _on_device);

template
col_major_matrix_ptr<mat_int_t>
make_col_major_matrix_ptr(
    const bool _on_device);

template
col_major_matrix_ptr<float>
make_col_major_matrix_ptr(
    const bool _on_device);

template
col_major_matrix_ptr<double>
make_col_major_matrix_ptr(
    const bool _on_device);

template
col_major_matrix_ptr<mat_int_t>
make_col_major_matrix_ptr(
    const mat_size_t _m,
    const mat_size_t _n,
    mat_int_t * _dense_val,
    const bool _on_device);

template
col_major_matrix_ptr<float>
make_col_major_matrix_ptr(
    const mat_size_t _m,
    const mat_size_t _n,
    float * _dense_val,
    const bool _on_device);

template
col_major_matrix_ptr<double>
make_col_major_matrix_ptr(
    const mat_size_t _m,
    const mat_size_t _n,
    double * _dense_val,
    const bool _on_device);

/* ************************************************************************** */

template
csr_matrix_ptr<mat_int_t>
make_csr_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const bool on_device);

template
csr_matrix_ptr<float>
make_csr_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const bool on_device);

template
csr_matrix_ptr<double>
make_csr_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const bool on_device);

template
csr_matrix_ptr<mat_int_t>
make_csr_matrix_ptr(
    const bool on_device);

template
csr_matrix_ptr<float>
make_csr_matrix_ptr(
    const bool on_device);

template
csr_matrix_ptr<double>
make_csr_matrix_ptr(
    const bool on_device);

template
csr_matrix_ptr<mat_int_t>
make_csr_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const mat_int_t * csr_row,
    const mat_int_t * csr_col,
    const mat_int_t * csr_val,
    const bool on_device);

template
csr_matrix_ptr<float>
make_csr_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const mat_int_t * csr_row,
    const mat_int_t * csr_col,
    const float * csr_val,
    const bool on_device);

template
csr_matrix_ptr<double>
make_csr_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const mat_int_t * csr_row,
    const mat_int_t * csr_col,
    const double * csr_val,
    const bool on_device);

/* ************************************************************************** */

template
coo_matrix_ptr<float>
make_coo_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const bool on_device);

template
coo_matrix_ptr<double>
make_coo_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const bool on_device);

template
coo_matrix_ptr<float>
make_coo_matrix_ptr(
    const bool on_device);

template
coo_matrix_ptr<double>
make_coo_matrix_ptr(
    const bool on_device);

template
coo_matrix_ptr<float>
make_coo_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const mat_int_t * coo_row,
    const mat_int_t * coo_col,
    const float * coo_val,
    const bool on_device);

template
coo_matrix_ptr<double>
make_coo_matrix_ptr(
    const mat_size_t m,
    const mat_size_t n,
    const mat_size_t nnz,
    const mat_int_t * coo_row,
    const mat_int_t * coo_col,
    const double * coo_val,
    const bool on_device);

/* ************************************************************************** */

template
dense_vector_ptr<float>
make_raw_dense_vector_ptr();

template
dense_vector_ptr<double>
make_raw_dense_vector_ptr();

template
dense_vector_ptr<char>
make_raw_dense_vector_ptr();

template
dense_vector_ptr<mat_int_t>
make_raw_dense_vector_ptr();

template
dense_vector_ptr<mat_size_t>
make_raw_dense_vector_ptr();

template
dense_vector_ptr<float>
make_raw_dense_vector_ptr(
    const mat_size_t,
    const bool on_device,
    float * dense_val);

template
dense_vector_ptr<double>
make_raw_dense_vector_ptr(
    const mat_size_t,
    const bool on_device,
    double * dense_val);

template
dense_vector_ptr<char>
make_raw_dense_vector_ptr(
    const mat_size_t,
    const bool on_device,
    char * dense_val);

template
dense_vector_ptr<mat_int_t>
make_raw_dense_vector_ptr(
    const mat_size_t,
    const bool on_device,
    mat_int_t * dense_val);

template
dense_vector_ptr<mat_size_t>
make_raw_dense_vector_ptr(
    const mat_size_t,
    const bool on_device,
    mat_size_t * dense_val);

template
dense_vector_ptr<float>
make_managed_dense_vector_ptr(
    const mat_size_t m,
    const bool on_device);

template
dense_vector_ptr<double>
make_managed_dense_vector_ptr(
    const mat_size_t m,
    const bool on_device);

template
dense_vector_ptr<char>
make_managed_dense_vector_ptr(
    const mat_size_t m,
    const bool on_device);

template
dense_vector_ptr<mat_int_t>
make_managed_dense_vector_ptr(
    const mat_size_t m,
    const bool on_device);

template
dense_vector_ptr<mat_size_t>
make_managed_dense_vector_ptr(
    const mat_size_t m,
    const bool on_device);

template
dense_vector_ptr<float>
make_managed_dense_vector_ptr(
    const bool on_device);

template
dense_vector_ptr<double>
make_managed_dense_vector_ptr(
    const bool on_device);

template
dense_vector_ptr<char>
make_managed_dense_vector_ptr(
    const bool on_device);

template
dense_vector_ptr<mat_int_t>
make_managed_dense_vector_ptr(
    const bool on_device);

template
dense_vector_ptr<mat_size_t>
make_managed_dense_vector_ptr(
    const bool on_device);

/**
 * *****************************************************************************
 * ******************************* GPU_HANDLE_T ********************************
 * *****************************************************************************
 */

gpu_handle_t::
gpu_handle_t()
{
    hipblasCreate(&cublas_handle);
    hipsparseCreate(&cusparse_handle);
    hipsolverDnCreate(&cusolver_handle);

    cublas_status = HIPBLAS_STATUS_SUCCESS;
    cusparse_status = HIPSPARSE_STATUS_SUCCESS;
    cusolver_status = HIPSOLVER_STATUS_SUCCESS;

    /* initialize to default stream */
    set_stream(hipStreamDefault);

    /* initialize to host scalar mode */
    set_scalar_mode(false);
}

/* ************************************************************************** */

gpu_handle_t::
~gpu_handle_t()
{
    hipblasDestroy(cublas_handle);
    hipsparseDestroy(cusparse_handle);
    hipsolverDnDestroy(cusolver_handle);
}

/* ************************************************************************** */

void
gpu_handle_t::
set_stream(
    const hipStream_t& stream)
{
    hipblasSetStream(cublas_handle, stream);
    hipsparseSetStream(cusparse_handle, stream);
    hipsolverSetStream(cusolver_handle, stream);

    m_stream = stream;
}

/* ************************************************************************** */

hipStream_t&
gpu_handle_t::
get_stream()
{
    return m_stream;
}

/* ************************************************************************** */

bool
gpu_handle_t::
get_scalar_mode()
{
    hipblasPointerMode_t cublas_mode;
    cublas_status = hipblasGetPointerMode(cublas_handle, &cublas_mode);

    hipsparsePointerMode_t cusparse_mode;
    cusparse_status = hipsparseGetPointerMode(cusparse_handle,
        &cusparse_mode);

    return ((cublas_mode == HIPBLAS_POINTER_MODE_DEVICE) &&
        (cusparse_mode == HIPSPARSE_POINTER_MODE_DEVICE));
}

/* ************************************************************************** */

void
gpu_handle_t::
set_scalar_mode(
    const bool scalar_device)
{
    cublas_status = hipblasSetPointerMode(cublas_handle,
        scalar_device ? HIPBLAS_POINTER_MODE_DEVICE :
        HIPBLAS_POINTER_MODE_HOST);
    cusparse_status = hipsparseSetPointerMode(cusparse_handle,
        scalar_device ? HIPSPARSE_POINTER_MODE_DEVICE :
        HIPSPARSE_POINTER_MODE_HOST);
}

/* ************************************************************************** */

void
gpu_handle_t::
push_scalar_mode()
{
    m_modes.push(get_scalar_mode());
}

/* ************************************************************************** */

void
gpu_handle_t::
pop_scalar_mode()
{
    if(!m_modes.empty())
    {
        set_scalar_mode(m_modes.top());
        m_modes.pop();
    }
}

/* ************************************************************************** */

void
gpu_handle_t::
__status_check(const char* s,
                const int f,
                const char* fname,
                const size_t line)
{
    if (f) {
        std::cerr << s << " (error " << f << ") at" << fname << ":" <<
            line << ", exiting..." << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

/* ************************************************************************** */

void
gpu_handle_t::
__cublas_check(
    const char* fname,
    const size_t line)
{
    if(cublas_status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "cuBLAS error " << cublas_err_str(cublas_status)
            << " at" << fname << ":" << line << ", exiting..." << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

/* ************************************************************************** */

void
gpu_handle_t::
__cusparse_check(
    const char* fname,
    const size_t line)
{
    if(cusparse_status != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cuSPARSE error " << cusparse_err_str(cusparse_status)
            << " at" << fname << ":" << line << ", exiting..." << std::endl;
            std::exit(EXIT_FAILURE);
    }
}

/* ************************************************************************** */

void
gpu_handle_t::
__cusolver_check(
    const char* fname,
    const size_t line)
{
    if(cusolver_status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cout << "cuSOLVER error " << cusolver_err_str(cusolver_status)
            << " at" << fname << ":" << line << ", exiting..." << std::endl;
            std::exit(EXIT_FAILURE);
    }
}

/* ************************************************************************** */

/**
    * Utility functions for error retrieval & evaluation.
    */
const char *
gpu_handle_t::
cublas_err_str(
    hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
        default:
            return "CUBLAS_UNKNOWN";
    }
}

/* ************************************************************************** */

const char *
gpu_handle_t::
cusparse_err_str(
    hipsparseStatus_t status)
{
    switch(status)
    {
        case HIPSPARSE_STATUS_SUCCESS:
            return "HIPSPARSE_STATUS_SUCCESS";
        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "HIPSPARSE_STATUS_NOT_INITIALIZED";
        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "HIPSPARSE_STATUS_ALLOC_FAILED";
        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "HIPSPARSE_STATUS_INVALID_VALUE";
        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "HIPSPARSE_STATUS_ARCH_MISMATCH";
        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "HIPSPARSE_STATUS_MAPPING_ERROR";
        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "HIPSPARSE_STATUS_EXECUTION_FAILED";
        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "HIPSPARSE_STATUS_INTERNAL_ERROR";
        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        default:
            return "CUSPARSE_UNKNOWN";
    }
}

/* ************************************************************************** */

const char *
gpu_handle_t::
cusolver_err_str(
    hipsolverStatus_t status)
{
    switch(status)
    {
        case HIPSOLVER_STATUS_SUCCESS:
            return "HIPSOLVER_STATUS_SUCCESS";
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "HIPSOLVER_STATUS_NOT_INITIALIZED";
        case HIPSOLVER_STATUS_ALLOC_FAILED:
            return "HIPSOLVER_STATUS_ALLOC_FAILED";
        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "HIPSOLVER_STATUS_INVALID_VALUE";
        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "HIPSOLVER_STATUS_ARCH_MISMATCH";
        case HIPSOLVER_STATUS_EXECUTION_FAILED:
            return "HIPSOLVER_STATUS_EXECUTION_FAILED";
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "HIPSOLVER_STATUS_INTERNAL_ERROR";
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        default:
            return "CUSOLVER_UNKNOWN";
    }
}

NS_CULIP_END
